
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

//val = 1 : initialise la matrice à 0,
//val = 0 : initialise la matrice avec des valeurs comprises entre 0-1,

void MatrixInit3D(float *M, int n, int p, int l,int val) {
    if(val==0){
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    M[i * p * n + j * n + k] = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
                    //M[i * p * n + j * n + k] = 1;
                }
            }
        }
    }
    else{
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    M[i * p * n + j * n + k] = 0;
                    }
                }
            }
    }
}



void MatrixPrint3D(float *M, int n, int p, int l) {
    for (int i = 0; i < l; ++i) {
        printf("Matrice %d :\n", i + 1);
        for (int j = 0; j < p; ++j) {
            for (int k = 0; k < n; ++k) {
                printf("%.2f\t", M[i * p * n + j * n + k]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout) {
    int n = gridDim.x;
    int p = gridDim.y;
    int l = gridDim.z;
    int size = n*p*l;
    int idx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    if (idx < size) {
        Mout[idx] = M1[idx]+M2[idx];
    }
}


__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int len) {
    int n = gridDim.x;
    int p = gridDim.y;

    int k = blockIdx.x;
    int j = blockIdx.y;

    if (k < n && j < p) {
        float sum = 0.0f;
        for (int h = 0; h < len; ++h) {
            sum += M1[j * p + h] * M2[h * n + k];
        }
        Mout[j * n + k] = sum;
    }
}

__device__ float activation_tanh(float M) {
    return tanh(M);
}

int main() {
    int n = 5;
    int p = 2;

    int n1 = 8;
    int p1 = 5;

    int nout = 8;
    int pout = 2;
    float *matrix;
    float *d_matrix; // Pointeur pour la matrice sur le GPU

    float *matrix2;
    float *d_matrix2;

    // float *matrixOut;
    // float *d_matrixOut;

    float *matrixOutMult;
    float *d_matrixOutMult;
    srand(time(NULL));

    // Allocation de mémoire pour la matrice sur le GPU
    hipMalloc((void **)&d_matrix, n * p * sizeof(float));
    hipMalloc((void **)&d_matrix2, n1 * p1 * sizeof(float));
    //cudaMalloc((void **)&d_matrixOut, n * p * sizeof(float));
    hipMalloc((void **)&d_matrixOutMult, nout * pout * sizeof(float));

    // Allocation et initialisation de la matrice sur le CPU
    matrix = (float *)malloc(n * p * sizeof(float));
    matrix2 = (float *)malloc(n1 * p1 * sizeof(float));
    //matrixOut = (float *)malloc(n * p * sizeof(float));
    matrixOutMult = (float *)malloc(nout * pout  * sizeof(float));

    MatrixInit3D(matrix, n, p,1,0);
    MatrixInit3D(matrix2, n1, p1,1,0);
    MatrixInit3D(matrixOutMult, nout, pout,1,0);

    // Copie de la matrice du CPU vers le GPU
    hipMemcpy(d_matrix, matrix, n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, n1 * p1 * sizeof(float), hipMemcpyHostToDevice);
    //dim3 gridDim(n,p);

    //cudaMatrixAdd<<<n,p>>>(d_matrix, d_matrix2, d_matrixOut, n, p);
    dim3 gridDim(nout,pout,1);
    cudaMatrixMult<<<gridDim,1>>>(d_matrix, d_matrix2, d_matrixOutMult, n);

    //cudaMemcpy(matrixOut, d_matrixOut, n * p * sizeof(float), cudaMemcpyDeviceToHost);
    hipMemcpy(matrixOutMult, d_matrixOutMult, nout*pout * sizeof(float), hipMemcpyDeviceToHost);

    // Affichage de la matrice sur le CPU
    printf("Matrice 1 :\n");

    MatrixPrint3D(matrix, n, p,1);

    printf("Matrice 2 :\n");

    MatrixPrint3D(matrix2, n1, p1,1);

    // printf("Somme :\n");

    // MatrixPrint(matrixOut, n, p);

    printf("Mult :\n");

    MatrixPrint3D(matrixOutMult, nout, pout,1);
    // Libération de la mémoire sur le CPU et le GPU
    free(matrix);
    hipFree(d_matrix);

    free(matrix2);
    hipFree(d_matrix2);

    // free(matrixOut);
    // cudaFree(d_matrixOut);

    free(matrixOutMult);
    hipFree(d_matrixOutMult);

    return 0;

}