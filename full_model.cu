
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

#define N 32
#define P 32
#define Q 6
#define K 5
#define WIDTH 28
#define HEIGHT 28
#define NO_IMG 5

void readFile(char* path, float * out){
    FILE *f = fopen(path, "r");

    if (f == NULL)
    {
        printf("Error: could not open file %s", path);
    }
    int i =0;

    while ((fscanf(f,"%f", &out[i])) != EOF){
        i++;
    }
    fclose(f);
}

void readImage(float * data, int no_img){
    FILE *fptr;
    unsigned int magic, nbImg, nbRows, nbCols;
    unsigned char val;

    //Open File
    if((fptr = fopen("train-images.idx3-ubyte","rb")) == NULL){
        printf("Can't open file");
        exit(1);
    }

    //Read File
    fread(&magic, sizeof(int), 1, fptr);
    fread(&nbImg, sizeof(int), 1, fptr);
    fread(&nbRows, sizeof(int), 1, fptr);
    fread(&nbCols, sizeof(int), 1, fptr);

    for(int k=0; k<no_img; k++){

        for(int i=2; i<WIDTH+2; i++){
            for(int j=2; j<HEIGHT+2; j++){ 
                fread(&val, sizeof(unsigned char), 1, fptr);  
                data[i*P+j]=(double)val/255;
            }
        }
    }  
}


//val = 1 : initialise la matrice à 0,
//val = 0 : initialise la matrice avec des valeurs comprises entre 0-1,

void MatrixInit3D(float *M, int n, int p, int l,int val) {
    if(val==0){
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    //M[i * p * n + j * n + k] = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
                    M[i * p * n + j * n + k] = 1;
                }
            }
        }
    }
    else{
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    M[i * p * n + j * n + k] = 0;
                    }
                }
            }
    }
}

void MatrixInit(float *M, int n, int p){
    for(int i=0; i<n; i++){
        for(int j=0; j<p;j++){
            M[i*p+j] = ((float)rand() / RAND_MAX)*2.0f - 1.0f;
        }
    }
}


void MatrixPrint(float *M, int n, int p){
    for(int i=0; i<n; i++){
        for(int j=0; j<p;j++){
            printf("%.2f\t", M[i*p+j]);
        }
        printf("\n");
    }
}

void MatrixPrint3D(float *M, int n, int p, int l) {
    for (int i = 0; i < l; ++i) {
        printf("Matrice %d :\n", i + 1);
        for (int j = 0; j < p; ++j) {
            for (int k = 0; k < n; ++k) {
                printf("%.2f\t", M[i * p * n + j * n + k]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


// Fonction d'addition de deux matrices sur CPU
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
        }
    }
}

// Fonction de multiplication de deux matrices NxN sur CPU
void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < n; ++k) {
                sum += M1[i * n + k] * M2[k * n + j];
            }
            Mout[i * n + j] = sum;
        }
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout) {
    int n = gridDim.x;
    int p = gridDim.y;
    int l = gridDim.z;
    int size = n*p*l;
    int idx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    if (idx < size) {
        Mout[idx] = M1[idx]+M2[idx];
    }
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int len) {
    int n = gridDim.x;
    int p = gridDim.y;

    int k = blockIdx.x;
    int j = blockIdx.y;

    if (k < n && j < p) {
        float sum = 0.0f;
        for (int h = 0; h < len; ++h) {
            sum += M1[j * p + h] * M2[h * n + k];
        }
        Mout[j * n + k] = sum;
    }
}
__device__ float activation_tanh(float M) {
    return tanh(M);
}


__global__ void Convolution2DGPU(float *input, float *kernels, float *output,int inputWidth, int kernelSize) {

    int n = gridDim.x;
    int p = gridDim.y;
    //int l = gridDim.z;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    float sum=0.0f;
    for (int ky = 0; ky < kernelSize; ++ky) {
        for (int kx = 0; kx < kernelSize; ++kx) {
            int inputX = x + kx;
            int inputY = y + ky;
            int inputIdx = inputY * inputWidth + inputX;
            int kernelIdx = z*kernelSize*kernelSize + ky*kernelSize+kx;
            sum += input[inputIdx] * kernels[kernelIdx];
        }
    }
    output[outputIdx] = activation_tanh(sum);
}

__global__ void Convolution3DGPU(float *input, float *kernels, float *output,int inputWidth, int kernelSize, int kerneldepth) {

    int n = gridDim.x;
    int p = gridDim.y;
    //int l = gridDim.z;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    float sum=0.0f;
    for(int kz = 0; kz < kerneldepth; ++kz){ 
        for (int ky = 0; ky < kernelSize; ++ky) {
            for (int kx = 0; kx < kernelSize; ++kx) {
                int inputX = x + kx;
                int inputY = y + ky;
                int inputIdx = inputY * inputWidth + inputX + kz*inputWidth*inputWidth;
                int kernelIdx = z*kernelSize*kernelSize*kerneldepth + kz*kernelSize*kernelSize + ky*kernelSize + kx;
                sum += input[inputIdx] * kernels[kernelIdx];
            }
        }
    } 
    output[outputIdx] = activation_tanh(sum);
}

__global__ void Moyennage2DGPU(float *input, float *output,int inputWidth) {

    int n = gridDim.x; //14
    int p = gridDim.y; //14
    //int n = 14; //14
    //int p = 14;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x; //pour parcourir mat sortie
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    float sum=0.0f;
    for (int ky = 0; ky < 2; ++ky) {
        for (int kx = 0; kx < 2; ++kx) {
            int inputX = 2*x + kx;
            int inputY = 2*y + ky;
            int inputIdx = z*inputWidth*inputWidth + inputY * inputWidth + inputX;
            sum += input[inputIdx];
        }
    }
    output[outputIdx] = sum/4;
}

__global__ void Flatten(float *input, float *output) {
    int n = gridDim.x;
    int p = gridDim.y;
    int l = gridDim.z;
    int size = n*p*l;
    int idx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    if (idx < size) {
        output[idx] = input[idx];
    }
}

/// modèle global

int main() {


    float *raw_data;
    float *d_raw_data; // Pointeur pour la matrice sur le GPU
    int nr=32;
    int pr=32;
    int lr=1;
    float *C1_data;
    float *d_C1_data;
    int nC1=28;
    int pC1=28;
    int lC1=6;
    float *S1_data;
    float *d_S1_data;
    int nS1=14;
    int pS1=14;
    int lS1=6;
    float *C1_kernel;
    float *d_C1_kernel;
    int nk=5;
    int pk=5;
    int lk=6;
    float *Conv2_data;
    float *d_Conv2_data;
    int nC2=10;
    int pC2=10;
    int lC2=16;
    float *S2_data;
    float *d_S2_data;
    int nS2=5;
    int pS2=5;
    int lS2=16;
    float *Conv2_kernel;
    float *d_Conv2_kernel;
    int nk2=5;
    int pk2=5;
    int lk2=16;
    int depthk2=6;
    float *flatten_data;
    float *d_flatten_data;
    int nf=400;
    int pf=1;
    int lf=1;

    //couche dense 1
    float *dense1_weight;
    float *d_dense1_weight;
    int ndw1=120;
    int pdw1=400;
    int ldw1=1;
    float *dense1_bias;
    float *d_dense1_bias;
    int ndb1=120;
    int pdb1=1;
    int ldb1=1;
    float *dense1_data;
    float *d_dense1_data;
    int ndd1=120;
    int pdd1=1;
    int ldd1=1;

    //couche dense 2
    float *dense2_weight;
    float *d_dense2_weight;
    int ndw2=84;
    int pdw2=120;
    int ldw2=1;
    float *dense2_bias;
    float *d_dense2_bias;
    int ndb2=84;
    int pdb2=1;
    int ldb2=1;
    float *dense2_data;
    float *d_dense2_data;
    int ndd2=84;
    int pdd2=1;
    int ldd2=1;

    //couche dense 3
    float *dense3_weight;
    float *d_dense3_weight;
    int ndw3=10;
    int pdw3=84;
    int ldw3=1;
    float *dense3_bias;
    float *d_dense3_bias;
    int ndb3=10;
    int pdb3=1;
    int ldb3=1;
    float *dense3_data;
    float *d_dense3_data;
    int ndd3=10;
    int pdd3=1;
    int ldd3=1;


    srand(time(NULL));

    // Allocation de mémoire pour la matrice sur le GPU
    hipMalloc((void **)&d_raw_data, nr * pr * lr*sizeof(float));
    hipMalloc((void **)&d_C1_data, nC1 * pC1 * lC1* sizeof(float));
    hipMalloc((void **)&d_S1_data, nS1 * pS1 * lS1* sizeof(float));
    hipMalloc((void **)&d_C1_kernel, nk * pk * lk* sizeof(float));
    hipMalloc((void **)&d_Conv2_data, nC2 * pC2 * lC2* sizeof(float));
    hipMalloc((void **)&d_S2_data, nS2 * pS2 * lS2* sizeof(float));
    hipMalloc((void **)&d_Conv2_kernel, nk2 * pk2 * lk2* depthk2* sizeof(float));
    hipMalloc((void **)&d_flatten_data, nf * pf * lf* sizeof(float));

    hipMalloc((void **)&d_dense1_weight, ndw1 * pdw1 * ldw1* sizeof(float));
    hipMalloc((void **)&d_dense2_weight, ndw2 * pdw2 * ldw2* sizeof(float));
    hipMalloc((void **)&d_dense3_weight, ndw3 * pdw3 * ldw3* sizeof(float));

    hipMalloc((void **)&d_dense1_bias, ndb1 * pdb1 * ldb1* sizeof(float));
    hipMalloc((void **)&d_dense2_bias, ndb2 * pdb2 * ldb2* sizeof(float));
    hipMalloc((void **)&d_dense3_bias, ndb3 * pdb3 * ldb3* sizeof(float));

    hipMalloc((void **)&d_dense1_data, ndd1 * pdd1 * ldd1* sizeof(float));
    hipMalloc((void **)&d_dense2_data, ndd2 * pdd2 * ldd2* sizeof(float));
    hipMalloc((void **)&d_dense3_data, ndd3 * pdd3 * ldd3* sizeof(float));

    // Allocation et initialisation de la matrice sur le CPU
    raw_data = (float *)malloc(nr * pr * lr*sizeof(float));
    
    C1_data = (float *)malloc(nC1 * pC1 * lC1*sizeof(float));
    S1_data = (float *)malloc(nS1 * pS1 * lS1*sizeof(float));
    C1_kernel = (float *)malloc(nk * pk * lk* sizeof(float));
    

    Conv2_data = (float *)malloc(nC2 * pC2 * lC2*sizeof(float));
    S2_data = (float *)malloc(nS2 * pS2 * lS2*sizeof(float));
    Conv2_kernel = (float *)malloc(nk2 * pk2 * lk2*depthk2* sizeof(float));
    flatten_data = (float *)malloc(nf * pf * lf* sizeof(float));

    dense1_weight = (float *)malloc(ndw1 * pdw1 * ldw1* sizeof(float));
    dense1_bias = (float *)malloc(ndb1 * pdb1 * ldb1* sizeof(float));
    dense1_data = (float *)malloc(ndd1 * pdd1 * ldd1* sizeof(float));

    dense2_weight = (float *)malloc(ndw2 * pdw2 * ldw2* sizeof(float));
    dense2_bias = (float *)malloc(ndb2 * pdb2 * ldb2* sizeof(float));
    dense2_data = (float *)malloc(ndd2 * pdd2 * ldd2* sizeof(float));

    dense3_weight = (float *)malloc(ndw3 * pdw3 * ldw3* sizeof(float));
    dense3_bias = (float *)malloc(ndb3 * pdb3 * ldb3* sizeof(float));
    dense3_data = (float *)malloc(ndd3 * pdd3 * ldd3* sizeof(float));

    MatrixInit3D(raw_data, nr,pr,lr,1);
    readImage(raw_data, 9);
    MatrixInit3D(C1_data, nC1,pC1,lC1,1);
    MatrixInit3D(S1_data, nS1,pS1,lS1,1);
    MatrixInit3D(C1_kernel, nk,pk,lk,1);
    readFile((char *)"weights/k1.h", C1_kernel);

    MatrixInit3D(Conv2_data, nC2,pC2,lC2,1);
    MatrixInit3D(S2_data, nS2,pS2,lS2,1);
    MatrixInit3D(Conv2_kernel, nk2,pk2,lk2*depthk2,1);
    readFile((char *)"weights/k2.h", Conv2_kernel);

    MatrixInit3D(flatten_data, nf,pf,lf,1);

    MatrixInit3D(dense1_weight, ndw1,pdw1,ldw1,1);
    readFile((char *)"weights/w1.h", dense1_weight);
    MatrixInit3D(dense1_bias, ndb1,pdb1,ldb1,1);
    readFile((char *)"weights/b1.h", dense1_bias);
    MatrixInit3D(dense1_data, ndd1,pdd1,ldd1,1);

    MatrixInit3D(dense2_weight, ndw2,pdw2,ldw2,1);
    readFile((char *)"weights/w2.h", dense2_weight);
    MatrixInit3D(dense2_bias, ndb2,pdb2,ldb2,1);
    readFile((char *)"weights/b2.h", dense2_bias);
    MatrixInit3D(dense2_data, ndd2,pdd2,ldd2,1);

    MatrixInit3D(dense3_weight, ndw3,pdw3,ldw3,1);
    readFile((char *)"weights/w3.h", dense3_weight);
    MatrixInit3D(dense3_bias, ndb3,pdb3,ldb3,1);
    readFile((char *)"weights/b3.h", dense3_bias);
    MatrixInit3D(dense3_data, ndd3,pdd3,ldd3,1);

    //Pour tester mes convolutions, création de kernels naifs : 
    // C1_kernel[12]=2;
    // C1_kernel[5*25+12]=1;

    // Conv2_kernel[12]=1;
    // Conv2_kernel[25+12]=1;
    // Conv2_kernel[92*25+12]=1;

    // Copie de la matrice du CPU vers le GPU
    hipMemcpy(d_raw_data, raw_data, nr * pr * lr*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_data, C1_data, nC1 * pC1 * lC1* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_S1_data, S1_data, nS1 * pS1 * lS1*  sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, nk * pk * lk* sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_Conv2_data, Conv2_data, nC2 * pC2 * lC2* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_S2_data, S2_data, nS2 * pS2 * lS2*  sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Conv2_kernel, Conv2_kernel, nk2 * pk2 * lk2*depthk2* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_flatten_data, flatten_data, nf * pf * lf* sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_dense1_weight, dense1_weight, ndw1 * pdw1 * ldw1* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense1_bias, dense1_bias, ndb1 * pdb1 * ldb1* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense1_data, dense1_data, ndd1 * pdd1 * ldd1* sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_dense2_weight, dense2_weight, ndw2 * pdw2 * ldw2* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense2_bias, dense2_bias, ndb2 * pdb2 * ldb2* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense2_data, dense2_data, ndd2 * pdd2 * ldd2* sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_dense3_weight, dense3_weight, ndw3 * pdw3 * ldw3* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense3_bias, dense3_bias, ndb3 * pdb3 * ldb3* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense3_data, dense3_data, ndd3 * pdd3 * ldd3* sizeof(float), hipMemcpyHostToDevice);

    dim3 gridDim(nC1,nC1,lC1);
    Convolution2DGPU<<<gridDim,1>>>(d_raw_data, d_C1_kernel, d_C1_data,nr,nk);

    dim3 gridDim2(nS1,pS1,lS1);
    Moyennage2DGPU<<<gridDim2,1>>>(d_C1_data, d_S1_data, nC1);

    dim3 gridDim3(nC2,nC2,lC2);
    Convolution3DGPU<<<gridDim3,1>>>(d_S1_data, d_Conv2_kernel, d_Conv2_data,nS1,nk2,depthk2);

    dim3 gridDim4(nS2,pS2,lS2);
    Moyennage2DGPU<<<gridDim4,1>>>(d_Conv2_data, d_S2_data, nC2);

    dim3 gridDim5(nS2,pS2,lS2);
    Flatten<<<gridDim5,1>>>(d_S2_data, d_flatten_data);

    dim3 gridDim6(ndd1,pdd1,ldd1);
    cudaMatrixMult<<<gridDim6,1>>>(d_flatten_data, d_dense1_weight, d_dense1_data, nf);

    dim3 gridDim7(ndd1,pdd1,ldd1);
    cudaMatrixAdd<<<gridDim7,1>>>(d_dense1_data, d_dense1_bias, d_dense1_data);

    dim3 gridDim8(ndd2,pdd2,ldd2);
    cudaMatrixMult<<<gridDim8,1>>>(d_dense1_data, d_dense2_weight, d_dense2_data, ndd1);

    dim3 gridDim9(ndd2,pdd2,ldd2);
    cudaMatrixAdd<<<gridDim9,1>>>(d_dense2_data, d_dense2_bias, d_dense2_data);

    dim3 gridDim10(ndd3,pdd3,ldd3);
    cudaMatrixMult<<<gridDim10,1>>>(d_dense2_data, d_dense3_weight, d_dense3_data, ndd2);

    dim3 gridDim11(ndd3,pdd3,ldd3);
    cudaMatrixAdd<<<gridDim11,1>>>(d_dense3_data, d_dense3_bias, d_dense3_data);

    hipMemcpy(raw_data, d_raw_data, nr * pr * lr* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C1_data, d_C1_data, nC1 * pC1 * lC1*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(S1_data, d_S1_data, nS1 * pS1 * lS1* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C1_kernel, d_C1_kernel, nk * pk * lk* sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(Conv2_data, d_Conv2_data, nC2 * pC2 * lC2*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(S2_data, d_S2_data, nS2 * pS2 * lS2* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Conv2_kernel, d_Conv2_kernel, nk2 * pk2 * lk2*depthk2* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(flatten_data, d_flatten_data, nf * pf * lf* sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(dense1_weight, d_dense1_weight, ndw1 * pdw1 * ldw1* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dense1_bias, d_dense1_bias, ndb1 * pdb1 * ldb1* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dense1_data, d_dense1_data, ndd1 * pdd1 * ldd1* sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(dense2_weight, d_dense2_weight, ndw2 * pdw2 * ldw2* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dense2_bias, d_dense2_bias, ndb2 * pdb2 * ldb2* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dense2_data, d_dense2_data, ndd2 * pdd2 * ldd2* sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(dense3_weight, d_dense3_weight, ndw3 * pdw3 * ldw3* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dense3_bias, d_dense3_bias, ndb3 * pdb3 * ldb3* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dense3_data, d_dense3_data, ndd3 * pdd3 * ldd3* sizeof(float), hipMemcpyDeviceToHost);


    // Affichage de la matrice sur le CPU
    // printf("Matrice raw_data :\n");

    // MatrixPrint3D(raw_data, nr, pr,lr);

    // printf("Matrice C1_data :\n");

    // MatrixPrint3D(C1_data, nC1, pC1,lC1);

    // printf("Matrice S1_data :\n");

    // MatrixPrint3D(S1_data, nS1, pS1,lS1);

    // printf("Matrice C1_kernel :\n");

    // MatrixPrint3D(C1_kernel, nk, pk,lk);

    // printf("Matrice Conv2_data :\n");

    // MatrixPrint3D(Conv2_data, nC2, pC2,lC2);

    // printf("Matrice S2_data :\n");

    // MatrixPrint3D(S2_data, nS2, pS2,lS2);

    // printf("Matrice Conv2_kernel :\n");

    // MatrixPrint3D(Conv2_kernel, nk2, pk2,lk2*depthk2);

    // printf("Matrice flatten_matrix :\n");

    // MatrixPrint3D(flatten_data, nf, pf,lf);

    // printf("Matrice dense1_weight :\n");

    // MatrixPrint3D(dense1_weight, ndw1, pdw1,ldw1);

    // printf("Matrice dense1_bias :\n");

    // MatrixPrint3D(dense1_bias, ndb1, pdb1,ldb1);

    // printf("Matrice dense1_data :\n");

    // MatrixPrint3D(dense1_data, ndd1, pdd1,ldd1);

    // printf("Matrice dense3_weight :\n");

    // MatrixPrint3D(dense3_weight, ndw3, pdw3,ldw3);

    // printf("Matrice dense3_bias :\n");

    // MatrixPrint3D(dense3_bias, ndb3, pdb3,ldb3);

    printf("Matrice dense3_data :\n");

    MatrixPrint3D(dense3_data, ndd3, pdd3,ldd3);

    // Libération de la mémoire sur le CPU et le GPU
    free(raw_data);
    hipFree(d_raw_data);

    free(C1_data);
    hipFree(d_C1_data);

    free(S1_data);
    hipFree(d_S1_data);

    free(C1_kernel);
    hipFree(d_C1_kernel);

    free(Conv2_data);
    hipFree(d_Conv2_data);

    free(S2_data);
    hipFree(d_S2_data);

    free(Conv2_kernel);
    hipFree(d_Conv2_kernel);

    free(flatten_data);
    hipFree(d_flatten_data);

    free(dense1_weight);
    hipFree(d_dense1_weight);
    free(dense2_weight);
    hipFree(d_dense2_weight);
    free(dense3_weight);
    hipFree(d_dense3_weight);
    
    free(dense1_bias);
    hipFree(d_dense1_bias);
    free(dense2_bias);
    hipFree(d_dense2_bias);
    free(dense3_bias);
    hipFree(d_dense3_bias);

    free(dense1_data);
    hipFree(d_dense1_data);
    free(dense2_data);
    hipFree(d_dense2_data);
    free(dense3_data);
    hipFree(d_dense3_data);

    return 0;

}

