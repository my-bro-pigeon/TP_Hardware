
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

#define N 32
#define P 32
#define Q 6
#define K 5
#define WIDTH 28
#define HEIGHT 28
#define NO_IMG 5

void readFile(char* path, double * out){
    FILE *f = fopen(path, "r");

    if (f == NULL)
    {
        printf("Error: could not open file %s", path);
    }
    int i =0;

    while ((fscanf(f,"%lf", &out[i])) != EOF){
        i++;
    }
    fclose(f);
}

void readImage(double * data, int no_img){
    FILE *fptr;
    unsigned int magic, nbImg, nbRows, nbCols;
    unsigned char val;

    //Open File
    if((fptr = fopen("train-images.idx3-ubyte","rb")) == NULL){
        printf("Can't open file");
        exit(1);
    }

    //Read File
    fread(&magic, sizeof(int), 1, fptr);
    fread(&nbImg, sizeof(int), 1, fptr);
    fread(&nbRows, sizeof(int), 1, fptr);
    fread(&nbCols, sizeof(int), 1, fptr);

    for(int k=0; k<no_img; k++){

        for(int i=2; i<WIDTH+2; i++){
            for(int j=2; j<HEIGHT+2; j++){ 
                fread(&val, sizeof(unsigned char), 1, fptr);  
                data[i*P+j]=(double)val/255;
            }
        }
    }  
}


//val = 1 : initialise la matrice à 0,
//val = 0 : initialise la matrice avec des valeurs comprises entre 0-1,

void MatrixInit3D(double *M, int n, int p, int l,int val) {
    if(val==0){
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    //M[i * p * n + j * n + k] = ((double)rand() / RAND_MAX) * 2.0f - 1.0f;
                    M[i * p * n + j * n + k] = 1;
                }
            }
        }
    }
    else{
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    M[i * p * n + j * n + k] = 0;
                    }
                }
            }
    }
}

void MatrixPrint3D(double *M, int n, int p, int l) {
    for (int i = 0; i < l; ++i) {
        printf("Matrice %d :\n", i + 1);
        for (int j = 0; j < p; ++j) {
            for (int k = 0; k < n; ++k) {
                printf("%lf\t", M[i * p * n + j * n + k]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


__global__ void cudaMatrixAdd(double *M1, double *M2, double *Mout) {
    int n = gridDim.x;
    int p = gridDim.y;
    int l = gridDim.z;
    int size = n*p*l;
    int idx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    if (idx < size) {
        Mout[idx] = M1[idx]+M2[idx];
    }
}

__global__ void cudaMatrixMult(double *M1, double *M2, double *Mout, int len) {
    int n = gridDim.x;
    int p = gridDim.y;

    int k = blockIdx.x;
    int j = blockIdx.y;

    if (k < n && j < p) {
        double sum = 0.0;
        for (int h = 0; h < len; ++h) {
            sum += M1[j * p + h] * M2[h * n + k];
        }
        Mout[j * n + k] = sum;
    }
}
__device__ double activation_tanh(double M) {
    return tanh(M);
}


__global__ void Convolution2DGPU(double *input, double *kernels, double *output,int inputWidth, int kernelSize) {

    int n = gridDim.x;
    int p = gridDim.y;
    //int l = gridDim.z;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    double sum=0.0f;
    for (int ky = 0; ky < kernelSize; ++ky) {
        for (int kx = 0; kx < kernelSize; ++kx) {
            int inputX = x + kx;
            int inputY = y + ky;
            int inputIdx = inputY * inputWidth + inputX;
            int kernelIdx = z*kernelSize*kernelSize + ky*kernelSize+kx;
            sum += input[inputIdx] * kernels[kernelIdx];
        }
    }
    output[outputIdx] = activation_tanh(sum);
}

__global__ void Convolution3DGPU(double *input, double *kernels, double *output,int inputWidth, int kernelSize, int kerneldepth) {

    int n = gridDim.x;
    int p = gridDim.y;
    //int l = gridDim.z;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    double sum=0.0f;
    for(int kz = 0; kz < kerneldepth; ++kz){ 
        for (int ky = 0; ky < kernelSize; ++ky) {
            for (int kx = 0; kx < kernelSize; ++kx) {
                int inputX = x + kx;
                int inputY = y + ky;
                int inputIdx = inputY * inputWidth + inputX + kz*inputWidth*inputWidth;
                int kernelIdx = z*kernelSize*kernelSize*kerneldepth + kz*kernelSize*kernelSize + ky*kernelSize + kx;
                sum += input[inputIdx] * kernels[kernelIdx];
            }
        }
    } 
    output[outputIdx] = activation_tanh(sum);
}

__global__ void Moyennage2DGPU(double *input, double *output,int inputWidth) {

    int n = gridDim.x; //14
    int p = gridDim.y; //14
    //int n = 14; //14
    //int p = 14;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x; //pour parcourir mat sortie
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    double sum=0.0f;
    for (int ky = 0; ky < 2; ++ky) {
        for (int kx = 0; kx < 2; ++kx) {
            int inputX = 2*x + kx;
            int inputY = 2*y + ky;
            int inputIdx = z*inputWidth*inputWidth + inputY * inputWidth + inputX;
            sum += input[inputIdx];
        }
    }
    output[outputIdx] = sum/4;
}

__global__ void Flatten(double *input, double *output) {
    int n = gridDim.x;
    int p = gridDim.y;
    int l = gridDim.z;
    int size = n*p*l;
    int idx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    if (idx < size) {
        output[idx] = input[idx];
    }
}

/// modèle global

int main() {


    double *raw_data;
    double *d_raw_data; // Pointeur pour la matrice sur le GPU
    int nr=32;
    int pr=32;
    int lr=1;
    double *C1_data;
    double *d_C1_data;
    int nC1=28;
    int pC1=28;
    int lC1=6;
    double *S1_data;
    double *d_S1_data;
    int nS1=14;
    int pS1=14;
    int lS1=6;
    double *C1_kernel;
    double *d_C1_kernel;
    int nk=5;
    int pk=5;
    int lk=6;
    double *Conv2_data;
    double *d_Conv2_data;
    int nC2=10;
    int pC2=10;
    int lC2=16;
    double *S2_data;
    double *d_S2_data;
    int nS2=5;
    int pS2=5;
    int lS2=16;
    double *Conv2_kernel;
    double *d_Conv2_kernel;
    int nk2=5;
    int pk2=5;
    int lk2=16;
    int depthk2=6;
    double *flatten_data;
    double *d_flatten_data;
    int nf=400;
    int pf=1;
    int lf=1;

    //couche dense 1
    double *dense1_weight;
    double *d_dense1_weight;
    int ndw1=120;
    int pdw1=400;
    int ldw1=1;
    double *dense1_bias;
    double *d_dense1_bias;
    int ndb1=120;
    int pdb1=1;
    int ldb1=1;
    double *dense1_data;
    double *d_dense1_data;
    int ndd1=120;
    int pdd1=1;
    int ldd1=1;

    //couche dense 2
    double *dense2_weight;
    double *d_dense2_weight;
    int ndw2=84;
    int pdw2=120;
    int ldw2=1;
    double *dense2_bias;
    double *d_dense2_bias;
    int ndb2=84;
    int pdb2=1;
    int ldb2=1;
    double *dense2_data;
    double *d_dense2_data;
    int ndd2=84;
    int pdd2=1;
    int ldd2=1;

    //couche dense 3
    double *dense3_weight;
    double *d_dense3_weight;
    int ndw3=10;
    int pdw3=84;
    int ldw3=1;
    double *dense3_bias;
    double *d_dense3_bias;
    int ndb3=10;
    int pdb3=1;
    int ldb3=1;
    double *dense3_data;
    double *d_dense3_data;
    int ndd3=10;
    int pdd3=1;
    int ldd3=1;


    srand(time(NULL));

    // Allocation de mémoire pour la matrice sur le GPU
    hipMalloc((void **)&d_raw_data, nr * pr * lr*sizeof(double));
    hipMalloc((void **)&d_C1_data, nC1 * pC1 * lC1* sizeof(double));
    hipMalloc((void **)&d_S1_data, nS1 * pS1 * lS1* sizeof(double));
    hipMalloc((void **)&d_C1_kernel, nk * pk * lk* sizeof(double));
    hipMalloc((void **)&d_Conv2_data, nC2 * pC2 * lC2* sizeof(double));
    hipMalloc((void **)&d_S2_data, nS2 * pS2 * lS2* sizeof(double));
    hipMalloc((void **)&d_Conv2_kernel, nk2 * pk2 * lk2* depthk2* sizeof(double));
    hipMalloc((void **)&d_flatten_data, nf * pf * lf* sizeof(double));

    hipMalloc((void **)&d_dense1_weight, ndw1 * pdw1 * ldw1* sizeof(double));
    hipMalloc((void **)&d_dense2_weight, ndw2 * pdw2 * ldw2* sizeof(double));
    hipMalloc((void **)&d_dense3_weight, ndw3 * pdw3 * ldw3* sizeof(double));

    hipMalloc((void **)&d_dense1_bias, ndb1 * pdb1 * ldb1* sizeof(double));
    hipMalloc((void **)&d_dense2_bias, ndb2 * pdb2 * ldb2* sizeof(double));
    hipMalloc((void **)&d_dense3_bias, ndb3 * pdb3 * ldb3* sizeof(double));

    hipMalloc((void **)&d_dense1_data, ndd1 * pdd1 * ldd1* sizeof(double));
    hipMalloc((void **)&d_dense2_data, ndd2 * pdd2 * ldd2* sizeof(double));
    hipMalloc((void **)&d_dense3_data, ndd3 * pdd3 * ldd3* sizeof(double));

    // Allocation et initialisation de la matrice sur le CPU
    raw_data = (double *)malloc(nr * pr * lr*sizeof(double));
    
    C1_data = (double *)malloc(nC1 * pC1 * lC1*sizeof(double));
    S1_data = (double *)malloc(nS1 * pS1 * lS1*sizeof(double));
    C1_kernel = (double *)malloc(nk * pk * lk* sizeof(double));
    

    Conv2_data = (double *)malloc(nC2 * pC2 * lC2*sizeof(double));
    S2_data = (double *)malloc(nS2 * pS2 * lS2*sizeof(double));
    Conv2_kernel = (double *)malloc(nk2 * pk2 * lk2*depthk2* sizeof(double));
    flatten_data = (double *)malloc(nf * pf * lf* sizeof(double));

    dense1_weight = (double *)malloc(ndw1 * pdw1 * ldw1* sizeof(double));
    dense1_bias = (double *)malloc(ndb1 * pdb1 * ldb1* sizeof(double));
    dense1_data = (double *)malloc(ndd1 * pdd1 * ldd1* sizeof(double));

    dense2_weight = (double *)malloc(ndw2 * pdw2 * ldw2* sizeof(double));
    dense2_bias = (double *)malloc(ndb2 * pdb2 * ldb2* sizeof(double));
    dense2_data = (double *)malloc(ndd2 * pdd2 * ldd2* sizeof(double));

    dense3_weight = (double *)malloc(ndw3 * pdw3 * ldw3* sizeof(double));
    dense3_bias = (double *)malloc(ndb3 * pdb3 * ldb3* sizeof(double));
    dense3_data = (double *)malloc(ndd3 * pdd3 * ldd3* sizeof(double));

    MatrixInit3D(raw_data, nr,pr,lr,1);
    readImage(raw_data, 5);
    MatrixInit3D(C1_data, nC1,pC1,lC1,1);
    MatrixInit3D(S1_data, nS1,pS1,lS1,1);
    MatrixInit3D(C1_kernel, nk,pk,lk,1);
    readFile((char *)"weights_nobias/k1.h", C1_kernel);

    MatrixInit3D(Conv2_data, nC2,pC2,lC2,1);
    MatrixInit3D(S2_data, nS2,pS2,lS2,1);
    MatrixInit3D(Conv2_kernel, nk2,pk2,lk2*depthk2,1);
    readFile((char *)"weights_nobias/k2.h", Conv2_kernel);

    MatrixInit3D(flatten_data, nf,pf,lf,1);

    MatrixInit3D(dense1_weight, ndw1,pdw1,ldw1,1);
    readFile((char *)"weights_nobias/w1.h", dense1_weight);
    MatrixInit3D(dense1_bias, ndb1,pdb1,ldb1,1);
    readFile((char *)"weights_nobias/b1.h", dense1_bias);
    MatrixInit3D(dense1_data, ndd1,pdd1,ldd1,1);

    MatrixInit3D(dense2_weight, ndw2,pdw2,ldw2,1);
    readFile((char *)"weights_nobias/w2.h", dense2_weight);
    MatrixInit3D(dense2_bias, ndb2,pdb2,ldb2,1);
    readFile((char *)"weights_nobias/b2.h", dense2_bias);
    MatrixInit3D(dense2_data, ndd2,pdd2,ldd2,1);

    MatrixInit3D(dense3_weight, ndw3,pdw3,ldw3,1);
    readFile((char *)"weights_nobias/w3.h", dense3_weight);
    MatrixInit3D(dense3_bias, ndb3,pdb3,ldb3,1);
    readFile((char *)"weights_nobias/b3.h", dense3_bias);
    MatrixInit3D(dense3_data, ndd3,pdd3,ldd3,1);

    //Pour tester mes convolutions, création de kernels naifs : 
    // C1_kernel[12]=2;
    // C1_kernel[5*25+12]=1;

    // Conv2_kernel[12]=1;
    // Conv2_kernel[25+12]=1;
    // Conv2_kernel[92*25+12]=1;

    // Copie de la matrice du CPU vers le GPU
    hipMemcpy(d_raw_data, raw_data, nr * pr * lr*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_data, C1_data, nC1 * pC1 * lC1* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_S1_data, S1_data, nS1 * pS1 * lS1*  sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, nk * pk * lk* sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_Conv2_data, Conv2_data, nC2 * pC2 * lC2* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_S2_data, S2_data, nS2 * pS2 * lS2*  sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Conv2_kernel, Conv2_kernel, nk2 * pk2 * lk2*depthk2* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_flatten_data, flatten_data, nf * pf * lf* sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_dense1_weight, dense1_weight, ndw1 * pdw1 * ldw1* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dense1_bias, dense1_bias, ndb1 * pdb1 * ldb1* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dense1_data, dense1_data, ndd1 * pdd1 * ldd1* sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_dense2_weight, dense2_weight, ndw2 * pdw2 * ldw2* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dense2_bias, dense2_bias, ndb2 * pdb2 * ldb2* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dense2_data, dense2_data, ndd2 * pdd2 * ldd2* sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_dense3_weight, dense3_weight, ndw3 * pdw3 * ldw3* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dense3_bias, dense3_bias, ndb3 * pdb3 * ldb3* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dense3_data, dense3_data, ndd3 * pdd3 * ldd3* sizeof(double), hipMemcpyHostToDevice);

    dim3 gridDim(nC1,nC1,lC1);
    Convolution2DGPU<<<gridDim,1>>>(d_raw_data, d_C1_kernel, d_C1_data,nr,nk);

    dim3 gridDim2(nS1,pS1,lS1);
    Moyennage2DGPU<<<gridDim2,1>>>(d_C1_data, d_S1_data, nC1);

    dim3 gridDim3(nC2,nC2,lC2);
    Convolution3DGPU<<<gridDim3,1>>>(d_S1_data, d_Conv2_kernel, d_Conv2_data,nS1,nk2,depthk2);

    dim3 gridDim4(nS2,pS2,lS2);
    Moyennage2DGPU<<<gridDim4,1>>>(d_Conv2_data, d_S2_data, nC2);

    dim3 gridDim5(nS2,pS2,lS2);
    Flatten<<<gridDim5,1>>>(d_S2_data, d_flatten_data);

    dim3 gridDim6(ndd1,pdd1,ldd1);
    cudaMatrixMult<<<gridDim6,1>>>(d_flatten_data, d_dense1_weight, d_dense1_data, nf);

    dim3 gridDim7(ndd1,pdd1,ldd1);
    cudaMatrixAdd<<<gridDim7,1>>>(d_dense1_data, d_dense1_bias, d_dense1_data);

    dim3 gridDim8(ndd2,pdd2,ldd2);
    cudaMatrixMult<<<gridDim8,1>>>(d_dense1_data, d_dense2_weight, d_dense2_data, ndd1);

    dim3 gridDim9(ndd2,pdd2,ldd2);
    cudaMatrixAdd<<<gridDim9,1>>>(d_dense2_data, d_dense2_bias, d_dense2_data);

    dim3 gridDim10(ndd3,pdd3,ldd3);
    cudaMatrixMult<<<gridDim10,1>>>(d_dense2_data, d_dense3_weight, d_dense3_data, ndd2);

    dim3 gridDim11(ndd3,pdd3,ldd3);
    cudaMatrixAdd<<<gridDim11,1>>>(d_dense3_data, d_dense3_bias, d_dense3_data);

    hipMemcpy(raw_data, d_raw_data, nr * pr * lr* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C1_data, d_C1_data, nC1 * pC1 * lC1*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(S1_data, d_S1_data, nS1 * pS1 * lS1* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C1_kernel, d_C1_kernel, nk * pk * lk* sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(Conv2_data, d_Conv2_data, nC2 * pC2 * lC2*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(S2_data, d_S2_data, nS2 * pS2 * lS2* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Conv2_kernel, d_Conv2_kernel, nk2 * pk2 * lk2*depthk2* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(flatten_data, d_flatten_data, nf * pf * lf* sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(dense1_weight, d_dense1_weight, ndw1 * pdw1 * ldw1* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dense1_bias, d_dense1_bias, ndb1 * pdb1 * ldb1* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dense1_data, d_dense1_data, ndd1 * pdd1 * ldd1* sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(dense2_weight, d_dense2_weight, ndw2 * pdw2 * ldw2* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dense2_bias, d_dense2_bias, ndb2 * pdb2 * ldb2* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dense2_data, d_dense2_data, ndd2 * pdd2 * ldd2* sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(dense3_weight, d_dense3_weight, ndw3 * pdw3 * ldw3* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dense3_bias, d_dense3_bias, ndb3 * pdb3 * ldb3* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dense3_data, d_dense3_data, ndd3 * pdd3 * ldd3* sizeof(double), hipMemcpyDeviceToHost);


    // Affichage de la matrice sur le CPU
    // printf("Matrice raw_data :\n");

    // MatrixPrint3D(raw_data, nr, pr,lr);

    // printf("Matrice C1_data :\n");

    // MatrixPrint3D(C1_data, nC1, pC1,lC1);

    // printf("Matrice S1_data :\n");

    // MatrixPrint3D(S1_data, nS1, pS1,lS1);

    // printf("Matrice C1_kernel :\n");

    // MatrixPrint3D(C1_kernel, nk, pk,lk);

    // printf("Matrice Conv2_data :\n");

    // MatrixPrint3D(Conv2_data, nC2, pC2,lC2);

    // printf("Matrice S2_data :\n");

    // MatrixPrint3D(S2_data, nS2, pS2,lS2);

    // printf("Matrice Conv2_kernel :\n");

    // MatrixPrint3D(Conv2_kernel, nk2, pk2,lk2*depthk2);

    // printf("Matrice flatten_matrix :\n");

    // MatrixPrint3D(flatten_data, nf, pf,lf);

    // printf("Matrice dense1_weight :\n");

    // MatrixPrint3D(dense1_weight, ndw1, pdw1,ldw1);

    // printf("Matrice dense1_bias :\n");

    // MatrixPrint3D(dense1_bias, ndb1, pdb1,ldb1);

    // printf("Matrice dense1_data :\n");

    // MatrixPrint3D(dense1_data, ndd1, pdd1,ldd1);

    // printf("Matrice dense3_weight :\n");

    // MatrixPrint3D(dense3_weight, ndw3, pdw3,ldw3);

    // printf("Matrice dense3_bias :\n");

    // MatrixPrint3D(dense3_bias, ndb3, pdb3,ldb3);

    printf("Matrice dense3_data :\n");

    MatrixPrint3D(dense3_data, ndd3, pdd3,ldd3);

    // Libération de la mémoire sur le CPU et le GPU
    free(raw_data);
    hipFree(d_raw_data);

    free(C1_data);
    hipFree(d_C1_data);

    free(S1_data);
    hipFree(d_S1_data);

    free(C1_kernel);
    hipFree(d_C1_kernel);

    free(Conv2_data);
    hipFree(d_Conv2_data);

    free(S2_data);
    hipFree(d_S2_data);

    free(Conv2_kernel);
    hipFree(d_Conv2_kernel);

    free(flatten_data);
    hipFree(d_flatten_data);

    free(dense1_weight);
    hipFree(d_dense1_weight);
    free(dense2_weight);
    hipFree(d_dense2_weight);
    free(dense3_weight);
    hipFree(d_dense3_weight);
    
    free(dense1_bias);
    hipFree(d_dense1_bias);
    free(dense2_bias);
    hipFree(d_dense2_bias);
    free(dense3_bias);
    hipFree(d_dense3_bias);

    free(dense1_data);
    hipFree(d_dense1_data);
    free(dense2_data);
    hipFree(d_dense2_data);
    free(dense3_data);
    hipFree(d_dense3_data);

    return 0;

}

