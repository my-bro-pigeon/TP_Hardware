#include "hip/hip_runtime.h"
/ TESTER LA PUISSANCE DU GPU ///

#include <stdlib.h>
#include <time.h>
#include <stdio.h>

//val = 1 : initialise la matrice à 0, 
//val = 0 : initialise la matrice avec des valeurs comprises entre 0-1, 

void MatrixInit3D(float *M, int n, int p, int l,int val) { 
    if(val==0){
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    //M[i * p * n + j * n + k] = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
                    M[i * p * n + j * n + k] = 1;
                }
            }
        }
    }
    else{
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < p; j++) {
                for (int k = 0; k < n; k++) {
                    M[i * p * n + j * n + k] = 0;
                    }
                }
            }
    }
}

void MatrixInit(float *M, int n, int p){    
    for(int i=0; i<n; i++){
        for(int j=0; j<p;j++){
            M[i*p+j] = ((float)rand() / RAND_MAX)*2.0f - 1.0f;           
        }
    }
}


void MatrixPrint(float *M, int n, int p){
    for(int i=0; i<n; i++){
        for(int j=0; j<p;j++){
            printf("%.2f\t", M[i*p+j]);
        }
        printf("\n");
    }
}

void MatrixPrint3D(float *M, int n, int p, int l) {
    for (int i = 0; i < l; ++i) {
        printf("Matrice %d :\n", i + 1);
        for (int j = 0; j < n; ++j) {
            for (int k = 0; k < p; ++k) {
                printf("%.2f\t", M[i * p * n + k * n + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


// Fonction d'addition de deux matrices sur CPU
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
        }
    }
}

// Fonction de multiplication de deux matrices NxN sur CPU
void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < n; ++k) {
                sum += M1[i * n + k] * M2[k * n + j];
            }
            Mout[i * n + j] = sum;
        }
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;

    if (i < n && j < p) {
        Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
    }
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += M1[row * n + k] * M2[k * n + col];
        }
        Mout[row * n + col] = sum;
    }
}

__device__ float activation_tanh(float M) {
    return tanh(M);
}


__global__ void Convolution2DGPU(float *input, float *kernels, float *output,int inputWidth, int kernelSize) { 

    int n = gridDim.x;
    int p = gridDim.y;
    //int l = gridDim.z;
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x;
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    float sum=0.0f;
    for (int ky = 0; ky < kernelSize; ++ky) {
        for (int kx = 0; kx < kernelSize; ++kx) {
            int inputX = x + kx;
            int inputY = y + ky;
            int inputIdx = inputY * inputWidth + inputX;
            int kernelIdx = z*kernelSize*kernelSize + ky*kernelSize+kx;
            sum += input[inputIdx] * kernels[kernelIdx];
        }
    }
    output[outputIdx] = activation_tanh(sum);
}
    

__global__ void Moyennage2DGPU(float *input, float *output,int inputWidth) { 

    int n = gridDim.x; //14
    int p = gridDim.y; //14
    //int n = 14; //14
    //int p = 14; 
    int outputIdx = blockIdx.z * n * p + blockIdx.y * n + blockIdx.x; //pour parcourir mat sortie
    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;

    float sum=0.0f;
    for (int ky = 0; ky < 2; ++ky) {
        for (int kx = 0; kx < 2; ++kx) {
            int inputX = 2*x + kx;
            int inputY = 2*y + ky;
            int inputIdx = z*inputWidth*inputWidth + inputY * inputWidth + inputX;
            sum += input[inputIdx];
        }
    }
    output[outputIdx] = sum/4;
}


int main() {
    int max_size = 10000; // Taille maximale de la matrice
    int step = 100; // Pas d'incrémentation de la taille de la matrice

    for (int n = step; n <= max_size; n += step) {
        float *h_matrix1, *h_matrix2, *h_result;
        float *d_matrix1, *d_matrix2, *d_result;

        // Allocation et initialisation des matrices sur CPU
        h_matrix1 = (float *)malloc(n * n * sizeof(float));
        h_matrix2 = (float *)malloc(n * n * sizeof(float));
        h_result = (float *)malloc(n * n * sizeof(float));

        // Initialisation des matrices sur CPU
        // ... (code pour l'initialisation des matrices h_matrix1 et h_matrix2)

        // Allocation de mémoire sur le GPU
        hipMalloc((void **)&d_matrix1, n * n * sizeof(float));
        hipMalloc((void **)&d_matrix2, n * n * sizeof(float));
        hipMalloc((void **)&d_result, n * n * sizeof(float));

        // Copie des données du CPU vers le GPU
        hipMemcpy(d_matrix1, h_matrix1, n * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, h_matrix2, n * n * sizeof(float), hipMemcpyHostToDevice);

        // Configuration des dimensions de la grille et des blocs pour la multiplication
        dim3 blockDim(16, 16); // Par exemple, 16x16 threads par bloc
        dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

        // Mesure du temps pour la multiplication sur GPU
        clock_t start = clock();
        cudaMatrixMult<<<gridDim, blockDim>>>(d_matrix1, d_matrix2, d_result, n);
        hipDeviceSynchronize(); // Attente de la fin du kernel
        clock_t end = clock();

        double time_spent = ((double)(end - start)) / CLOCKS_PER_SEC;
        printf("Taille de la matrice : %d x %d, Temps écoulé : %.5f secondes\n", n, n, time_spent);

        // Libération de la mémoire sur le GPU
        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_result);

        // Libération de la mémoire sur le CPU
        free(h_matrix1);
        free(h_matrix2);
        free(h_result);
    }

    return 0;
}
